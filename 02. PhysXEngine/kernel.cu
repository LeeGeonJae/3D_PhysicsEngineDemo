
#include <hip/hip_runtime.h>
//#include "type.h"
//
//#include <stdio.h>
//#include <cuda_runtime.h>
//#include <device_launch_parameters.h>
//#include <directxtk\SimpleMath.h>
//#include <xlocale>
//
//__device__ float2 Sub(const float2& lhs, const float2& rhs)
//{
//    float2 result;
//    result.x = lhs.x - rhs.x;
//    result.y = lhs.y - rhs.y;
//    return result;
//}
//
//__device__ float3 Sub(const float3& lhs, const float3& rhs)
//{
//    float3 result;
//    result.x = lhs.x - rhs.x;
//    result.y = lhs.y - rhs.y;
//    result.z = lhs.z - rhs.z;
//    return result;
//}
//
//__device__ float3 cross(const float3& a, const float3& b) {
//    return {
//        a.y * b.z - a.z * b.y,
//        a.z * b.x - a.x * b.z,
//        a.x * b.y - a.y * b.x
//    };
//}
//
//__device__ float DotProduct(const float3& a, const float3& b) {
//    return a.x * b.x + a.y * b.y + a.z * b.z;
//}
//
//__device__ void NormalizeVector(float3& vectors)
//{
//    float length = vectors.x + vectors.y + vectors.z;
//    if (length > 0) {
//        vectors.x /= length;
//        vectors.y /= length;
//        vectors.z /= length;
//    }
//}
//
//// CUDA Ŀ�� �Լ� ����
//__global__ void CalculateNormals(
//    float3* vertices,
//    float2* uvs,
//    unsigned int vertexSize,
//    unsigned int* indices,
//    unsigned int indexSize,
//    PhysicsEngine::PhysicsVertex* buffer)
//{
//    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//    if (idx >= indexSize / 3) return;
//
//    unsigned int i0 = indices[idx * 3];
//    unsigned int i1 = indices[idx * 3 + 1];
//    unsigned int i2 = indices[idx * 3 + 2];
//
//    float3 v0 = vertices[i0];
//    float3 v1 = vertices[i1];
//    float3 v2 = vertices[i2];
//
//    float2 uv0 = uvs[i0];
//    float2 uv1 = uvs[i1];
//    float2 uv2 = uvs[i2];
//
//    float3 edge1 = Sub(v1, v0);
//    float3 edge2 = Sub(v2, v0);
//
//    float2 deltaUV1 = Sub(uv1, uv0);
//    float2 deltaUV2 = Sub(uv2, uv0);
//
//    float f = 1.0f / (deltaUV1.x * deltaUV2.y - deltaUV2.x * deltaUV1.y);
//
//    float3 tangent;
//    tangent.x = f * (deltaUV2.y * edge1.x - deltaUV1.y * edge2.x);
//    tangent.y = f * (deltaUV2.y * edge1.y - deltaUV1.y * edge2.y);
//    tangent.z = f * (deltaUV2.y * edge1.z - deltaUV1.y * edge2.z);
//    NormalizeVector(tangent);
//
//    float3 bitangent;
//    bitangent.x = f * (-deltaUV2.x * edge1.x + deltaUV1.x * edge2.x);
//    bitangent.y = f * (-deltaUV2.x * edge1.y + deltaUV1.x * edge2.y);
//    bitangent.z = f * (-deltaUV2.x * edge1.z + deltaUV1.x * edge2.z);
//    NormalizeVector(bitangent);
//
//    float3 normal = cross(edge1, edge2);
//    NormalizeVector(normal);
//
//    buffer[i0].normal.x += normal.x;
//    buffer[i0].normal.y += normal.y;
//    buffer[i0].normal.z += normal.z;
//
//    buffer[i1].normal.x += normal.x;
//    buffer[i1].normal.y += normal.y;
//    buffer[i1].normal.z += normal.z;
//
//    buffer[i2].normal.x += normal.x;
//    buffer[i2].normal.y += normal.y;
//    buffer[i2].normal.z += normal.z;
//
//    buffer[i0].tangent.x += tangent.x;
//    buffer[i0].tangent.y += tangent.y;
//    buffer[i0].tangent.z += tangent.z;
//
//    buffer[i1].tangent.x += tangent.x;
//    buffer[i1].tangent.y += tangent.y;
//    buffer[i1].tangent.z += tangent.z;
//
//    buffer[i2].tangent.x += tangent.x;
//    buffer[i2].tangent.y += tangent.y;
//    buffer[i2].tangent.z += tangent.z;
//
//    buffer[i0].biTangent.x += bitangent.x;
//    buffer[i0].biTangent.y += bitangent.y;
//    buffer[i0].biTangent.z += bitangent.z;
//
//    buffer[i1].biTangent.x += bitangent.x;
//    buffer[i1].biTangent.y += bitangent.y;
//    buffer[i1].biTangent.z += bitangent.z;
//
//    buffer[i2].biTangent.x += bitangent.x;
//    buffer[i2].biTangent.y += bitangent.y;
//    buffer[i2].biTangent.z += bitangent.z;
//}
//
//using namespace PhysicsEngine;
//
//// Ŀ�� ȣ���� ���� C++ �Լ� ����
//extern "C" void launchCalculateNormals(
//    float3 * vertices,
//    float2 * uvs,
//    unsigned int vertexSize,
//    unsigned int* indices,
//    unsigned int indexSize,
//    PhysicsEngine::PhysicsVertex * buffer)
//{
//    int threadsPerBlock = 256;
//    int blocksPerGrid = (indexSize / 3 + threadsPerBlock - 1) / threadsPerBlock;
//
//    CalculateNormals<<<blocksPerGrid, threadsPerBlock >>>(
//        vertices, uvs, vertexSize, indices, indexSize, buffer);
//
//    cudaDeviceSynchronize();
//}