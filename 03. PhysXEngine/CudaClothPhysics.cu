#include "hip/hip_runtime.h"
#include "CudaClothPhysics.h"
#include "cudamanager\PxCudaContext.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#include <directxtk\SimpleMath.h>

#include "EngineDataConverter.h"

#pragma comment (lib, "cudart.lib")

__device__ physx::PxVec2 Sub(const physx::PxVec2& lhs, const physx::PxVec2& rhs) {
	return { lhs.x - rhs.x, lhs.y - rhs.y };
}

__device__ physx::PxVec4 Sub(const physx::PxVec4& lhs, const physx::PxVec4& rhs) {
	return { lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z, lhs.w - rhs.w };
}

__device__ physx::PxVec3 cross(const physx::PxVec4& a, const physx::PxVec4& b) {
	return { a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x };
}

__device__ float DotProduct(const physx::PxVec3& a, const physx::PxVec3& b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ void NormalizeVector(physx::PxVec3& vec) {
	float length = sqrtf(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
	if (length > 0) {
		vec.x /= length;
		vec.y /= length;
		vec.z /= length;
	}
}

// CUDA Ŀ�� �Լ� ����
__global__ void CalculateNormals(
	physx::PxVec4* vertices,
	physx::PxVec2* uvs,
    unsigned int vertexSize,
    unsigned int* indices,
    unsigned int indexSize,
    physics::PhysicsVertex* buffer)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= indexSize / 3) return;

    unsigned int i0 = indices[idx * 3];
    unsigned int i1 = indices[idx * 3 + 1];
    unsigned int i2 = indices[idx * 3 + 2];

    physx::PxVec4 v0 = vertices[i0];
    physx::PxVec4 v1 = vertices[i1];
    physx::PxVec4 v2 = vertices[i2];
	v0.z = -v0.z;
	v1.z = -v1.z;
	v2.z = -v2.z;

    physx::PxVec2 uv0 = uvs[i0];
    physx::PxVec2 uv1 = uvs[i1];
    physx::PxVec2 uv2 = uvs[i2];

    physx::PxVec4 edge1 = Sub(v1, v0);
    physx::PxVec4 edge2 = Sub(v2, v0);

	physx::PxVec2 deltaUV1 = Sub(uv1, uv0);
	physx::PxVec2 deltaUV2 = Sub(uv2, uv0);

    float f = 1.0f / (deltaUV1.x * deltaUV2.y - deltaUV2.x * deltaUV1.y);

	physx::PxVec3 tangent;
    tangent.x = f * (deltaUV2.y * edge1.x - deltaUV1.y * edge2.x);
    tangent.y = f * (deltaUV2.y * edge1.y - deltaUV1.y * edge2.y);
    tangent.z = f * (deltaUV2.y * edge1.z - deltaUV1.y * edge2.z);
    NormalizeVector(tangent);

	physx::PxVec3 bitangent;
    bitangent.x = f * (-deltaUV2.x * edge1.x + deltaUV1.x * edge2.x);
    bitangent.y = f * (-deltaUV2.x * edge1.y + deltaUV1.x * edge2.y);
    bitangent.z = f * (-deltaUV2.x * edge1.z + deltaUV1.x * edge2.z);
    NormalizeVector(bitangent);

	physx::PxVec3 normal = cross(edge1, edge2);
    NormalizeVector(normal);

	buffer[i0].position.x = v0.x;
	buffer[i0].position.y = v0.y;
	buffer[i0].position.z = v0.z;
	buffer[i1].position.x = v1.x;
	buffer[i1].position.y = v1.y;
	buffer[i1].position.z = v1.z;
	buffer[i2].position.x = v2.x;
	buffer[i2].position.y = v2.y;
	buffer[i2].position.z = v2.z;

	buffer[i0].normal.x = normal.x;
	buffer[i0].normal.y = normal.y;
	buffer[i0].normal.z = normal.z;
					    
	buffer[i1].normal.x = normal.x;
	buffer[i1].normal.y = normal.y;
	buffer[i1].normal.z = normal.z;
					    
	buffer[i2].normal.x = normal.x;
	buffer[i2].normal.y = normal.y;
	buffer[i2].normal.z = normal.z;

	buffer[i0].tangent.x = tangent.x;
	buffer[i0].tangent.y = tangent.y;
	buffer[i0].tangent.z = tangent.z;
						 
	buffer[i1].tangent.x = tangent.x;
	buffer[i1].tangent.y = tangent.y;
	buffer[i1].tangent.z = tangent.z;
						 
	buffer[i2].tangent.x = tangent.x;
	buffer[i2].tangent.y = tangent.y;
	buffer[i2].tangent.z = tangent.z;

	buffer[i0].biTangent.x = bitangent.x;
	buffer[i0].biTangent.y = bitangent.y;
	buffer[i0].biTangent.z = bitangent.z;
						   
	buffer[i1].biTangent.x = bitangent.x;
	buffer[i1].biTangent.y = bitangent.y;
	buffer[i1].biTangent.z = bitangent.z;
						   
	buffer[i2].biTangent.x = bitangent.x;
	buffer[i2].biTangent.y = bitangent.y;
	buffer[i2].biTangent.z = bitangent.z;
}

__global__ void processVerticesKernel(unsigned int* sameVerticesFirst, unsigned int* sameVerticesSecond,
	physics::PhysicsVertex* buffer, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) {
		int first = sameVerticesFirst[idx];
		int second = sameVerticesSecond[idx];

		// �߰����� ����Ͽ� ���� ������ �и�
		float3 normalTemp, tangentTemp, biTangentTemp;

		normalTemp.x = buffer[first].normal.x + buffer[second].normal.x;
		normalTemp.y = buffer[first].normal.y + buffer[second].normal.y;
		normalTemp.z = buffer[first].normal.z + buffer[second].normal.z;

		tangentTemp.x = buffer[first].tangent.x + buffer[second].tangent.x;
		tangentTemp.y = buffer[first].tangent.y + buffer[second].tangent.y;
		tangentTemp.z = buffer[first].tangent.z + buffer[second].tangent.z;

		biTangentTemp.x = buffer[first].biTangent.x + buffer[second].biTangent.x;
		biTangentTemp.y = buffer[first].biTangent.y + buffer[second].biTangent.y;
		biTangentTemp.z = buffer[first].biTangent.z + buffer[second].biTangent.z;

		buffer[first].normal.x = normalTemp.x;
		buffer[first].normal.y = normalTemp.y;
		buffer[first].normal.z = normalTemp.z;
		buffer[second].normal.x = normalTemp.x;
		buffer[second].normal.y = normalTemp.y;
		buffer[second].normal.z = normalTemp.z;

		buffer[first].tangent.x = tangentTemp.x;
		buffer[first].tangent.y = tangentTemp.y;
		buffer[first].tangent.z = tangentTemp.z;
		buffer[second].tangent.x = tangentTemp.x;
		buffer[second].tangent.y = tangentTemp.y;
		buffer[second].tangent.z = tangentTemp.z;

		buffer[first].biTangent.x = biTangentTemp.x;
		buffer[first].biTangent.y = biTangentTemp.y;
		buffer[first].biTangent.z = biTangentTemp.z;
		buffer[second].biTangent.x = biTangentTemp.x;
		buffer[second].biTangent.y = biTangentTemp.y;
		buffer[second].biTangent.z = biTangentTemp.z;
	}
}

struct SimpleVector3 {
	float x, y, z;
};

struct SimpleMatrix {
	float m[4][4];
};

__device__ SimpleVector3 multiply(SimpleMatrix& mat, const SimpleVector3& vec)
{
	SimpleVector3 result;
	result.x = mat.m[0][0] * vec.x + mat.m[1][0] * vec.y + mat.m[2][0] * vec.z + mat.m[3][0] * 1.0f;
	result.y = mat.m[0][1] * vec.x + mat.m[1][1] * vec.y + mat.m[2][1] * vec.z + mat.m[3][1] * 1.0f;
	result.z = mat.m[0][2] * vec.x + mat.m[1][2] * vec.y + mat.m[2][2] * vec.z + mat.m[3][2] * 1.0f;
	return result;
}

// Ŀ�� �Լ�
__global__ void TransformVertices(physx::PxVec4* particle, SimpleMatrix previousTransformInverse, SimpleMatrix newTransform, int vertexCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= vertexCount) return;

	SimpleVector3 vertex;
	vertex.x = particle[idx].x;
	vertex.y = particle[idx].y;
	vertex.z = -particle[idx].z;

	// ���� worldTransform�� ����� ����
	vertex = multiply(previousTransformInverse, vertex);

	// ���ο� worldTransform ����
	vertex = multiply(newTransform, vertex);

	// ��ȯ�� vertex ����
	particle[idx].x = vertex.x;
	particle[idx].y = vertex.y;
	particle[idx].z = -vertex.z;
}

namespace physics
{
	CudaClothPhysics::CudaClothPhysics(unsigned int id, unsigned int layerNumber)
		: mID(id)
		, mLayNumber(layerNumber)
		, mTotalClothMass()
		, mWorldTransform()
		, mSprings()
		, mPBDMaterial(nullptr)
		, mParticleSystem(nullptr)
		, mClothBuffer(nullptr)
		, mClothBufferHelper(nullptr)
		, mPhase(nullptr)
		, mPositionInvMass(nullptr)
		, mVelocity(nullptr)
		, mVertices()
		, mIndices()
	{
	}
	CudaClothPhysics::~CudaClothPhysics()
	{
	}

	bool CudaClothPhysics::Initialize(const PhysicsClothInfo& info, physx::PxPhysics* physics, physx::PxScene* scene, physx::PxCudaContextManager* cudaContextManager)
	{
		int deviceCount;
		hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);
		if (cudaStatus != hipSuccess || deviceCount == 0) {
			std::cerr << "CUDA ����̽� �ʱ�ȭ ����" << std::endl;
			return false;
		}

		cudaStatus = hipSetDevice(0); // ù ��° CUDA ����̽� ���
		if (cudaStatus != hipSuccess) {
			std::cerr << "CUDA ����̽� ���� ����" << std::endl;
			return false;
		}

		if (cudaContextManager == nullptr)
			return false;

		settingInfoData(info);
		extractSpringsData(info.indices, info.indexSize);

		// ���(Material) ����
		mPBDMaterial = physics->createPBDMaterial(
			info.materialInfo.friction,
			info.materialInfo.damping,
			info.materialInfo.adhesion,
			info.materialInfo.viscosity,
			info.materialInfo.vorticityConfinement,
			info.materialInfo.surfaceTension,
			info.materialInfo.cohesion,
			info.materialInfo.lift,
			info.materialInfo.drag,
			info.materialInfo.cflCoefficient,
			info.materialInfo.gravityScale);

		createClothParticle(physics, scene, cudaContextManager);

		return true;
	}

	bool areVerticesEqual(const DirectX::SimpleMath::Vector3& vertex1, const DirectX::SimpleMath::Vector3& vertex2, float epsilon = 1e-6) {
		return (std::abs(vertex1.x - vertex2.x) < epsilon) &&
			(std::abs(vertex1.y - vertex2.y) < epsilon) &&
			(std::abs(vertex1.z - vertex2.z) < epsilon);
	}

	void CudaClothPhysics::extractSpringsData(unsigned int* indices, unsigned int indexSize)
	{
		// �ﰢ�� ������ �ε����� ��ȸ
		for (size_t i = 0; i < indexSize; i += 3)
		{
			unsigned int v1 = indices[i];
			unsigned int v2 = indices[i + 1];
			unsigned int v3 = indices[i + 2];

			// ���ĵ� ������ vertex ���� �߰��Ͽ� �ߺ��� ���� ����
			auto addEdge = [this](unsigned int a, unsigned int b)
				{
					if (a > b) std::swap(a, b);
					mSprings.insert({ a, b });
				};

			addEdge(v1, v2);
			addEdge(v2, v3);
			addEdge(v3, v1);
		}

		mSameVertices.reserve(mVertices.size() / 3);
		for (int i = 0; i < mVertices.size(); i++)
		{
			for (int j = i + 1; j < mVertices.size(); j++)
			{
				if (areVerticesEqual(mVertices[i], mVertices[j]))
				{
					mSprings.insert({ i, j });
					mSameVertices.push_back({ i, j });
				}
			}
		}
	}

	void CudaClothPhysics::settingInfoData(const PhysicsClothInfo& info)
	{
		RegisterD3D11VertexBufferWithCUDA((ID3D11Buffer*)info.vertexBuffer);
		RegisterD3D11IndexBufferWithCUDA((ID3D11Buffer*)info.indexBuffer);

		mWorldTransform = info.worldTransform;
		mTotalClothMass = info.totalClothMass;

		mIndices.resize(info.indexSize);
		memcpy(mIndices.data(), info.indices, info.indexSize * sizeof(unsigned int));

		mUV.resize(info.vertexSize);
		mVertices.resize(info.vertexSize);

		for (int i = 0; i < info.vertexSize; i++)
		{
			mVertices[i].x = info.vertices[i].x;
			mVertices[i].y = info.vertices[i].y;
			mVertices[i].z = info.vertices[i].z;
			mUV[i] = info.uv[i];
		}

		for (auto& vertex : mVertices)
		{
			MulMatrixVector3(mWorldTransform, vertex);
		}

		for (int i = 0; i < info.vertexSize; i++)
		{
			mVertices[i].z = -mVertices[i].z;
		}
	}

	void CudaClothPhysics::createClothParticle(physx::PxPhysics* physics, physx::PxScene* scene, physx::PxCudaContextManager* cudaContextManager)
	{
		// ���� �� ������, �ﰢ���� ���� ���
		const physx::PxU32 numParticles = mVertices.size();	// ���� ����
		const physx::PxU32 numSprings = mSprings.size();	// ���� �ϳ��� �̿��ϴ� ���ڵ鿡 ������ ���� �����µ�, �� ������ ����
		const physx::PxU32 numTriangles = mIndices.size() / 3;	// �ﰢ�� ����

		// ���� �ý����� ����
		const physx::PxReal particleMass = mTotalClothMass / mVertices.size();
		const physx::PxReal restOffset = 2.f;

		// ���� �ý��� ����
		mParticleSystem = physics->createPBDParticleSystem(*cudaContextManager);

		mParticleSystem->setRestOffset(1.f);
		mParticleSystem->setContactOffset(restOffset + 0.02f);
		mParticleSystem->setParticleContactOffset(restOffset + 0.02f);
		mParticleSystem->setSolidRestOffset(restOffset);

		// ���� ���� �ý��� �߰�
		scene->addActor(*mParticleSystem);

		// ������ ���¸� �����ϴ� ���� ����
		const physx::PxU32 particlePhase = mParticleSystem->createPhase(mPBDMaterial, physx::PxParticlePhaseFlags(
			physx::PxParticlePhaseFlag::eParticlePhaseSelfCollideFilter | physx::PxParticlePhaseFlag::eParticlePhaseSelfCollide));

		mClothBufferHelper = physx::ExtGpu::PxCreateParticleClothBufferHelper(1, numTriangles, numSprings, numParticles, cudaContextManager);
		// ����, ������ �ﰢ���� ���¸� �����ϱ� ���� ���� �Ҵ�
		mPhase = cudaContextManager->allocPinnedHostBuffer<physx::PxU32>(numParticles);
		mPositionInvMass = cudaContextManager->allocPinnedHostBuffer<physx::PxVec4>(numParticles);
		mVelocity = cudaContextManager->allocPinnedHostBuffer<physx::PxVec4>(numParticles);

		settingParticleBuffer(numSprings, numTriangles, numParticles, particlePhase, particleMass);
		createCloth(numParticles, cudaContextManager);
	}

	float calculateVectorMagnitude(const DirectX::SimpleMath::Vector3& point1, const DirectX::SimpleMath::Vector3& point2) {
		float dx = point2.x - point1.x;
		float dy = point2.y - point1.y;
		float dz = point2.z - point1.z;

		return std::sqrt(dx * dx + dy * dy + dz * dz);
	}

	void CudaClothPhysics::settingParticleBuffer(
		const physx::PxU32& numSprings,
		const physx::PxU32& numTriangles,
		const physx::PxU32& numParticles,
		const physx::PxU32& particlePhase,
		const physx::PxReal& particleMass)
	{
		const physx::PxReal stretchStiffness = 100.f;
		const physx::PxReal shearStiffness = 100.f;
		const physx::PxReal springDamping = 0.1f;

		// ��ƼŬ ������ �� Ʈ���̾ޱ� ����
		physx::PxArray<physx::PxParticleSpring> springs;
		springs.reserve(numSprings);
		physx::PxArray<physx::PxU32> triangles;
		triangles.reserve(numTriangles * 3);

		// ���� ���� ����
		for (int i = 0; i < numParticles; i++)
		{
			mPositionInvMass[i] = physx::PxVec4(mVertices[i].x, mVertices[i].y + 300.f, mVertices[i].z, 1.f / particleMass);
			mPhase[i] = particlePhase;
			mVelocity[i] = physx::PxVec4(0.f);
		}

		// ������ �߰�
		for (auto line : mSprings)
		{
			physx::PxParticleSpring spring = { line.first, line.second, calculateVectorMagnitude(mVertices[line.first], mVertices[line.second]), stretchStiffness, springDamping, 0 };
			springs.pushBack(spring);
		}

		// �ﰢ�� �߰�
		for (int i = 0; i < mIndices.size(); i += 3)
		{
			triangles.pushBack(mIndices[i]);
			triangles.pushBack(mIndices[i + 1]);
			triangles.pushBack(mIndices[i + 2]);
		}

		// ������ ������ �� �ﰢ�� ���� ������ �����Ǿ����� Ȯ��
		PX_ASSERT(numSprings == springs.size());
		PX_ASSERT(numTriangles == triangles.size() / 3);

		// õ���� ���ۿ� ������ �߰�
		mClothBufferHelper->addCloth(0.f, 0.f, 0.f, triangles.begin(), numTriangles, springs.begin(), numSprings, mPositionInvMass, numParticles);
	}

	void CudaClothPhysics::createCloth(const physx::PxU32& numParticles, physx::PxCudaContextManager* cudaContextManager)
	{
		// ������ ���¸� ��Ÿ���� ���� ����
		physx::ExtGpu::PxParticleBufferDesc bufferDesc;
		bufferDesc.maxParticles = numParticles;
		bufferDesc.numActiveParticles = numParticles;
		bufferDesc.positions = mPositionInvMass;
		bufferDesc.velocities = mVelocity;
		bufferDesc.phases = mPhase;

		// õ���� ���� ��������
		const physx::PxParticleClothDesc& clothDesc = mClothBufferHelper->getParticleClothDesc();

		// ���� õ���� ��ó���� ����
		physx::PxParticleClothPreProcessor* clothPreProcessor = PxCreateParticleClothPreProcessor(cudaContextManager);

		// ���� õ�� ���� �� ó��
		physx::PxPartitionedParticleCloth output;
		clothPreProcessor->partitionSprings(clothDesc, output);
		clothPreProcessor->release();

		// õ�� ���� ����
		mClothBuffer = physx::ExtGpu::PxCreateAndPopulateParticleClothBuffer(bufferDesc, clothDesc, output, cudaContextManager);
		mParticleSystem->addParticleBuffer(mClothBuffer);

		// ���� ����
		mClothBufferHelper->release();

		// �Ҵ�� �޸� ����
		//cudaContextManager->freePinnedHostBuffer(mPositionInvMass);
		cudaContextManager->freePinnedHostBuffer(mVelocity);
		cudaContextManager->freePinnedHostBuffer(mPhase);
	}

	bool CudaClothPhysics::RegisterD3D11VertexBufferWithCUDA(ID3D11Buffer* buffer)
	{
		hipError_t cudaStatus = cudaGraphicsD3D11RegisterResource(&mCudaVertexResource, buffer, hipGraphicsRegisterFlagsNone);
		if (cudaStatus != hipSuccess)
		{
			std::cerr << "Direct3D ���ҽ� ��� ����" << std::endl;
			return false;
		}
		return true;
	}

	bool CudaClothPhysics::RegisterD3D11IndexBufferWithCUDA(ID3D11Buffer* buffer)
	{
		hipError_t cudaStatus = cudaGraphicsD3D11RegisterResource(&mCudaIndexResource, buffer, hipGraphicsRegisterFlagsNone);
		if (cudaStatus != hipSuccess)
		{
			std::cerr << "Direct3D ���ҽ� ��� ����" << std::endl;
			return false;
		}
		return true;
	}

	bool CudaClothPhysics::UpdatePhysicsCloth(physx::PxCudaContextManager* cudaContextManager)
	{
		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &mCudaVertexResource);
		//if (cudaStatus != hipSuccess) {
		//	std::cerr << "hipGraphicsMapResources ����: " << hipGetErrorString(cudaStatus) << std::endl;
		//	return false;
		//}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t size = 0;
		hipGraphicsResourceGetMappedPointer(&devPtr, &size, mCudaVertexResource);
		//if (cudaStatus != hipSuccess) {
		//	std::cerr << "hipGraphicsResourceGetMappedPointer ����: " << hipGetErrorString(cudaStatus) << std::endl;
		//	return false;
		//}

		unsigned int deviceVertexSize = mVertices.size();
		unsigned int deviceIndexSize = mIndices.size();

		physx::PxVec2* d_uvs;
		unsigned int* d_indices;

		hipMalloc(&d_uvs, mUV.size() * sizeof(DirectX::SimpleMath::Vector2));
		hipMalloc(&d_indices, mIndices.size() * sizeof(unsigned int));
		hipMemcpy(d_uvs, mUV.data(), mUV.size() * sizeof(DirectX::SimpleMath::Vector2), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_indices, mIndices.data(), mIndices.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);


		int particleSize = mClothBuffer->getNbActiveParticles();
		physx::PxVec4* particle = mClothBuffer->getPositionInvMasses();

		int threadsPerBlock = 256;
		int blocksPerGrid = (mIndices.size() / 3 + threadsPerBlock - 1) / threadsPerBlock;

		// CUDA �Լ� ����
		CalculateNormals <<<blocksPerGrid, threadsPerBlock>>> (
			particle, d_uvs, deviceVertexSize, d_indices, deviceIndexSize, (PhysicsVertex*)devPtr);

		blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

		std::vector<unsigned int> firstVertex;
		std::vector<unsigned int> secondVertex;
		firstVertex.resize(mSameVertices.size());
		secondVertex.resize(mSameVertices.size());
		for (int i = 0; i < mSameVertices.size(); i++)
		{
			firstVertex.push_back(mSameVertices[i].first);
			secondVertex.push_back(mSameVertices[i].second);
		}

		unsigned int* d_firstVertex;
		unsigned int* d_secondVertex;

		hipMalloc(&d_firstVertex, firstVertex.size() * sizeof(unsigned int));
		hipMalloc(&d_secondVertex, secondVertex.size() * sizeof(unsigned int));
		hipMemcpy(d_firstVertex, firstVertex.data(), firstVertex.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_secondVertex, secondVertex.data(), secondVertex.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);

		processVerticesKernel << <blocksPerGrid, threadsPerBlock >> > (d_firstVertex, d_secondVertex, (PhysicsVertex*)devPtr, mVertices.size());

		hipDeviceSynchronize();

		// CUDA ���ҽ��� �����
		hipGraphicsUnmapResources(1, &mCudaVertexResource);

		// �޸� ����
		hipFree(d_uvs);
		hipFree(d_indices);
		hipFree(d_firstVertex);
		hipFree(d_secondVertex);

		return false;
	}

	void CudaClothPhysics::GetPhysicsCloth(PhysicsClothGetData& data)
	{
		data.worldTransform = mWorldTransform;
	}

	void CudaClothPhysics::SetPhysicsCloth(const PhysicsClothSetData& data)
	{
		DirectX::SimpleMath::Matrix prevTransform = mWorldTransform.Invert();
		DirectX::SimpleMath::Matrix nextTransform = data.worldTransform;

		SimpleMatrix prevMatrix;
		SimpleMatrix nextMatrix;

		prevMatrix.m[0][0] = prevTransform._11;
		prevMatrix.m[0][1] = prevTransform._12;
		prevMatrix.m[0][2] = prevTransform._13;
		prevMatrix.m[0][3] = prevTransform._14;
		prevMatrix.m[1][0] = prevTransform._21;
		prevMatrix.m[1][1] = prevTransform._22;
		prevMatrix.m[1][2] = prevTransform._23;
		prevMatrix.m[1][3] = prevTransform._24;
		prevMatrix.m[2][0] = prevTransform._31;
		prevMatrix.m[2][1] = prevTransform._32;
		prevMatrix.m[2][2] = prevTransform._33;
		prevMatrix.m[2][3] = prevTransform._34;
		prevMatrix.m[3][0] = prevTransform._41;
		prevMatrix.m[3][1] = prevTransform._42;
		prevMatrix.m[3][2] = prevTransform._43;
		prevMatrix.m[3][3] = prevTransform._44;

		nextMatrix.m[0][0] = nextTransform._11;
		nextMatrix.m[0][1] = nextTransform._12;
		nextMatrix.m[0][2] = nextTransform._13;
		nextMatrix.m[0][3] = nextTransform._14;
		nextMatrix.m[1][0] = nextTransform._21;
		nextMatrix.m[1][1] = nextTransform._22;
		nextMatrix.m[1][2] = nextTransform._23;
		nextMatrix.m[1][3] = nextTransform._24;
		nextMatrix.m[2][0] = nextTransform._31;
		nextMatrix.m[2][1] = nextTransform._32;
		nextMatrix.m[2][2] = nextTransform._33;
		nextMatrix.m[2][3] = nextTransform._34;
		nextMatrix.m[3][0] = nextTransform._41;
		nextMatrix.m[3][1] = nextTransform._42;
		nextMatrix.m[3][2] = nextTransform._43;
		nextMatrix.m[3][3] = nextTransform._44;

		physx::PxVec4* particle = mClothBuffer->getPositionInvMasses();

		int threadsPerBlock = 256;
		int blocksPerGrid = (mVertices.size() + threadsPerBlock - 1) / threadsPerBlock;

		TransformVertices <<<blocksPerGrid, threadsPerBlock>>>(particle, prevMatrix, nextMatrix, mVertices.size());

		mClothBuffer->raiseFlags(physx::PxParticleBufferFlag::eUPDATE_POSITION);

		mWorldTransform = data.worldTransform;
	}
}
