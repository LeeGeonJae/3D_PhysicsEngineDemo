#include "hip/hip_runtime.h"
#include "CudaClothPhysics.h"
#include "cudamanager\PxCudaContext.h"

#include <iostream>

#pragma comment (lib, "cudart.lib")

__device__ physx::PxVec2 Sub(const physx::PxVec2& lhs, const physx::PxVec2& rhs)
{
	physx::PxVec2 result;
    result.x = lhs.x - rhs.x;
    result.y = lhs.y - rhs.y;
    return result;
}

__device__ physx::PxVec4 Sub(const physx::PxVec4& lhs, const physx::PxVec4& rhs)
{
	physx::PxVec4 result;
    result.x = lhs.x - rhs.x;
    result.y = lhs.y - rhs.y;
    result.z = lhs.z - rhs.z;
	result.w = lhs.w - rhs.w;
    return result;
}

__device__ physx::PxVec3 cross(const physx::PxVec4& a, const physx::PxVec4& b) {
    return {
        a.y * b.z - a.z * b.y,
        a.z * b.x - a.x * b.z,
        a.x * b.y - a.y * b.x,
    };
}

__device__ float DotProduct(const physx::PxVec3& a, const physx::PxVec3& b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ void NormalizeVector(physx::PxVec3& vectors)
{
    float length = vectors.x + vectors.y + vectors.z;
    if (length > 0) {
        vectors.x /= length;
        vectors.y /= length;
        vectors.z /= length;
    }
}

// CUDA Ŀ�� �Լ� ����
__global__ void CalculateNormals(
	physx::PxVec4* vertices,
	physx::PxVec2* uvs,
    unsigned int vertexSize,
    unsigned int* indices,
    unsigned int indexSize,
    physics::PhysicsVertex* buffer)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= indexSize / 3) return;

    unsigned int i0 = indices[idx * 3];
    unsigned int i1 = indices[idx * 3 + 1];
    unsigned int i2 = indices[idx * 3 + 2];

    physx::PxVec4 v0 = vertices[i0];
    physx::PxVec4 v1 = vertices[i1];
    physx::PxVec4 v2 = vertices[i2];
	v0.z = -v0.z;
	v1.z = -v1.z;
	v2.z = -v2.z;

    physx::PxVec2 uv0 = uvs[i0];
    physx::PxVec2 uv1 = uvs[i1];
    physx::PxVec2 uv2 = uvs[i2];

    physx::PxVec4 edge1 = Sub(v1, v0);
    physx::PxVec4 edge2 = Sub(v2, v0);

	physx::PxVec2 deltaUV1 = Sub(uv1, uv0);
	physx::PxVec2 deltaUV2 = Sub(uv2, uv0);

    float f = 1.0f / (deltaUV1.x * deltaUV2.y - deltaUV2.x * deltaUV1.y);

	physx::PxVec3 tangent;
    tangent.x = f * (deltaUV2.y * edge1.x - deltaUV1.y * edge2.x);
    tangent.y = f * (deltaUV2.y * edge1.y - deltaUV1.y * edge2.y);
    tangent.z = f * (deltaUV2.y * edge1.z - deltaUV1.y * edge2.z);
    NormalizeVector(tangent);

	physx::PxVec3 bitangent;
    bitangent.x = f * (-deltaUV2.x * edge1.x + deltaUV1.x * edge2.x);
    bitangent.y = f * (-deltaUV2.x * edge1.y + deltaUV1.x * edge2.y);
    bitangent.z = f * (-deltaUV2.x * edge1.z + deltaUV1.x * edge2.z);
    NormalizeVector(bitangent);

	physx::PxVec3 normal = cross(edge1, edge2);
    NormalizeVector(normal);

	buffer[i0].position.x = v0.x;
	buffer[i0].position.y = v0.y;
	buffer[i0].position.z = v0.z;

	buffer[i1].position.x = v1.x;
	buffer[i1].position.y = v1.y;
	buffer[i1].position.z = v1.z;

	buffer[i2].position.x = v2.x;
	buffer[i2].position.y = v2.y;
	buffer[i2].position.z = v2.z;

    buffer[i0].normal.x += normal.x;
    buffer[i0].normal.y += normal.y;
    buffer[i0].normal.z += normal.z;

    buffer[i1].normal.x += normal.x;
    buffer[i1].normal.y += normal.y;
    buffer[i1].normal.z += normal.z;

    buffer[i2].normal.x += normal.x;
    buffer[i2].normal.y += normal.y;
    buffer[i2].normal.z += normal.z;

    buffer[i0].tangent.x += tangent.x;
    buffer[i0].tangent.y += tangent.y;
    buffer[i0].tangent.z += tangent.z;

    buffer[i1].tangent.x += tangent.x;
    buffer[i1].tangent.y += tangent.y;
    buffer[i1].tangent.z += tangent.z;

    buffer[i2].tangent.x += tangent.x;
    buffer[i2].tangent.y += tangent.y;
    buffer[i2].tangent.z += tangent.z;

    buffer[i0].biTangent.x += bitangent.x;
    buffer[i0].biTangent.y += bitangent.y;
    buffer[i0].biTangent.z += bitangent.z;

    buffer[i1].biTangent.x += bitangent.x;
    buffer[i1].biTangent.y += bitangent.y;
    buffer[i1].biTangent.z += bitangent.z;

    buffer[i2].biTangent.x += bitangent.x;
    buffer[i2].biTangent.y += bitangent.y;
    buffer[i2].biTangent.z += bitangent.z;
}

namespace physics
{
	CudaClothPhysics::CudaClothPhysics(unsigned int id, unsigned int layerNumber)
		: mID(id)
		, mLayNumber(layerNumber)
		, mTotalClothMass()
		, mWorldTransform()
		, mSprings()
		, mPBDMaterial(nullptr)
		, mParticleSystem(nullptr)
		, mClothBuffer(nullptr)
		, mClothBufferHelper(nullptr)
		, mPhase(nullptr)
		, mPositionInvMass(nullptr)
		, mVelocity(nullptr)
		, mVertices()
		, mIndices()
	{
	}
	CudaClothPhysics::~CudaClothPhysics()
	{
	}

	bool CudaClothPhysics::Initialize(const PhysicsClothInfo& info, physx::PxPhysics* physics, physx::PxScene* scene, physx::PxCudaContextManager* cudaContextManager)
	{
		int deviceCount;
		hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);
		if (cudaStatus != hipSuccess || deviceCount == 0) {
			std::cerr << "CUDA ����̽� �ʱ�ȭ ����" << std::endl;
			return false;
		}

		cudaStatus = hipSetDevice(0); // ù ��° CUDA ����̽� ���
		if (cudaStatus != hipSuccess) {
			std::cerr << "CUDA ����̽� ���� ����" << std::endl;
			return false;
		}

		if (cudaContextManager == nullptr)
			return false;

		settingInfoData(info);
		extractSpringsData(info.indices, info.indexSize);

		// ���(Material) ����
		mPBDMaterial = physics->createPBDMaterial(
			info.materialInfo.friction,
			info.materialInfo.damping,
			info.materialInfo.adhesion,
			info.materialInfo.viscosity,
			info.materialInfo.vorticityConfinement,
			info.materialInfo.surfaceTension,
			info.materialInfo.cohesion,
			info.materialInfo.lift,
			info.materialInfo.drag,
			info.materialInfo.cflCoefficient,
			info.materialInfo.gravityScale);

		createClothParticle(physics, scene, cudaContextManager);

		return true;
	}

	bool areVerticesEqual(const DirectX::SimpleMath::Vector3& vertex1, const DirectX::SimpleMath::Vector3& vertex2, float epsilon = 1e-6) {
		return (std::abs(vertex1.x - vertex2.x) < epsilon) &&
			(std::abs(vertex1.y - vertex2.y) < epsilon) &&
			(std::abs(vertex1.z - vertex2.z) < epsilon);
	}

	void CudaClothPhysics::extractSpringsData(unsigned int* indices, unsigned int indexSize)
	{
		// �ﰢ�� ������ �ε����� ��ȸ
		for (size_t i = 0; i < indexSize; i += 3)
		{
			unsigned int v1 = indices[i];
			unsigned int v2 = indices[i + 1];
			unsigned int v3 = indices[i + 2];

			// ���ĵ� ������ vertex ���� �߰��Ͽ� �ߺ��� ���� ����
			auto addEdge = [this](unsigned int a, unsigned int b)
				{
					if (a > b) std::swap(a, b);
					mSprings.insert({ a, b });
				};

			addEdge(v1, v2);
			addEdge(v2, v3);
			addEdge(v3, v1);
		}

		mSameVertices.reserve(mVertices.size() / 3);
		for (int i = 0; i < mVertices.size(); i++)
		{
			for (int j = i + 1; j < mVertices.size(); j++)
			{
				if (areVerticesEqual(mVertices[i], mVertices[j]))
				{
					mSprings.insert({ i, j });
					mSameVertices.push_back({ i, j });
				}
			}
		}
	}

	void CudaClothPhysics::settingInfoData(const PhysicsClothInfo& info)
	{
		mWorldTransform = info.worldTransform;
		mTotalClothMass = info.totalClothMass;

		mIndices.resize(info.indexSize);
		memcpy(mIndices.data(), info.indices, info.indexSize * sizeof(unsigned int));

		mVertices.resize(info.vertexSize);
		mUV.resize(info.vertexSize);
		for (int i = 0; i < info.vertexSize; i++)
		{
			mVertices[i].x = info.vertices[i].x;
			mVertices[i].y = info.vertices[i].y;
			mVertices[i].z = -info.vertices[i].z;
			mUV[i] = info.uv[i];
		}
	}

	void CudaClothPhysics::createClothParticle(physx::PxPhysics* physics, physx::PxScene* scene, physx::PxCudaContextManager* cudaContextManager)
	{
		// ���� �� ������, �ﰢ���� ���� ���
		const physx::PxU32 numParticles = mVertices.size();	// ���� ����
		const physx::PxU32 numSprings = mSprings.size();	// ���� �ϳ��� �̿��ϴ� ���ڵ鿡 ������ ���� �����µ�, �� ������ ����
		const physx::PxU32 numTriangles = mIndices.size() / 3;	// �ﰢ�� ����

		// ���� �ý����� ����
		const physx::PxReal particleMass = mTotalClothMass / mVertices.size();
		const physx::PxReal restOffset = 2.f;

		// ���� �ý��� ����
		mParticleSystem = physics->createPBDParticleSystem(*cudaContextManager);

		mParticleSystem->setRestOffset(1.f);
		mParticleSystem->setContactOffset(restOffset + 0.02f);
		mParticleSystem->setParticleContactOffset(restOffset + 0.02f);
		mParticleSystem->setSolidRestOffset(restOffset);

		// ���� ���� �ý��� �߰�
		scene->addActor(*mParticleSystem);

		// ������ ���¸� �����ϴ� ���� ����
		const physx::PxU32 particlePhase = mParticleSystem->createPhase(mPBDMaterial, physx::PxParticlePhaseFlags(
			physx::PxParticlePhaseFlag::eParticlePhaseSelfCollideFilter | physx::PxParticlePhaseFlag::eParticlePhaseSelfCollide));

		mClothBufferHelper = physx::ExtGpu::PxCreateParticleClothBufferHelper(1, numTriangles, numSprings, numParticles, cudaContextManager);
		// ����, ������ �ﰢ���� ���¸� �����ϱ� ���� ���� �Ҵ�
		mPhase = cudaContextManager->allocPinnedHostBuffer<physx::PxU32>(numParticles);
		mPositionInvMass = cudaContextManager->allocPinnedHostBuffer<physx::PxVec4>(numParticles);
		mVelocity = cudaContextManager->allocPinnedHostBuffer<physx::PxVec4>(numParticles);

		settingParticleBuffer(numSprings, numTriangles, numParticles, particlePhase, particleMass);
		createCloth(numParticles, cudaContextManager);
	}

	float calculateVectorMagnitude(const DirectX::SimpleMath::Vector3& point1, const DirectX::SimpleMath::Vector3& point2) {
		float dx = point2.x - point1.x;
		float dy = point2.y - point1.y;
		float dz = point2.z - point1.z;

		return std::sqrt(dx * dx + dy * dy + dz * dz);
	}

	void CudaClothPhysics::settingParticleBuffer(
		const physx::PxU32& numSprings,
		const physx::PxU32& numTriangles,
		const physx::PxU32& numParticles,
		const physx::PxU32& particlePhase,
		const physx::PxReal& particleMass)
	{
		const physx::PxReal stretchStiffness = 100.f;
		const physx::PxReal shearStiffness = 100.f;
		const physx::PxReal springDamping = 0.1f;

		// ��ƼŬ ������ �� Ʈ���̾ޱ� ����
		physx::PxArray<physx::PxParticleSpring> springs;
		springs.reserve(numSprings);
		physx::PxArray<physx::PxU32> triangles;
		triangles.reserve(numTriangles * 3);

		// ���� ���� ����
		for (int i = 0; i < numParticles; i++)
		{
			mPositionInvMass[i] = physx::PxVec4(mVertices[i].x, mVertices[i].y + 300.f, mVertices[i].z, 1.f / particleMass);
			mPhase[i] = particlePhase;
			mVelocity[i] = physx::PxVec4(0.f);
		}

		// ������ �߰�
		for (auto line : mSprings)
		{
			physx::PxParticleSpring spring = { line.first, line.second, calculateVectorMagnitude(mVertices[line.first], mVertices[line.second]), stretchStiffness, springDamping, 0 };
			springs.pushBack(spring);
		}

		// �ﰢ�� �߰�
		for (int i = 0; i < mIndices.size(); i += 3)
		{
			triangles.pushBack(mIndices[i]);
			triangles.pushBack(mIndices[i + 1]);
			triangles.pushBack(mIndices[i + 2]);
		}

		// ������ ������ �� �ﰢ�� ���� ������ �����Ǿ����� Ȯ��
		PX_ASSERT(numSprings == springs.size());
		PX_ASSERT(numTriangles == triangles.size() / 3);

		// õ���� ���ۿ� ������ �߰�
		mClothBufferHelper->addCloth(0.f, 0.f, 0.f, triangles.begin(), numTriangles, springs.begin(), numSprings, mPositionInvMass, numParticles);
	}

	void CudaClothPhysics::createCloth(const physx::PxU32& numParticles, physx::PxCudaContextManager* cudaContextManager)
	{
		// ������ ���¸� ��Ÿ���� ���� ����
		physx::ExtGpu::PxParticleBufferDesc bufferDesc;
		bufferDesc.maxParticles = numParticles;
		bufferDesc.numActiveParticles = numParticles;
		bufferDesc.positions = mPositionInvMass;
		bufferDesc.velocities = mVelocity;
		bufferDesc.phases = mPhase;

		// õ���� ���� ��������
		const physx::PxParticleClothDesc& clothDesc = mClothBufferHelper->getParticleClothDesc();

		// ���� õ���� ��ó���� ����
		physx::PxParticleClothPreProcessor* clothPreProcessor = PxCreateParticleClothPreProcessor(cudaContextManager);

		// ���� õ�� ���� �� ó��
		physx::PxPartitionedParticleCloth output;
		clothPreProcessor->partitionSprings(clothDesc, output);
		clothPreProcessor->release();

		// õ�� ���� ����
		mClothBuffer = physx::ExtGpu::PxCreateAndPopulateParticleClothBuffer(bufferDesc, clothDesc, output, cudaContextManager);
		mParticleSystem->addParticleBuffer(mClothBuffer);

		// ���� ����
		mClothBufferHelper->release();

		// �Ҵ�� �޸� ����
		//cudaContextManager->freePinnedHostBuffer(mPositionInvMass);
		cudaContextManager->freePinnedHostBuffer(mVelocity);
		cudaContextManager->freePinnedHostBuffer(mPhase);
	}

	bool CudaClothPhysics::RegisterD3D11BufferWithCUDA(ID3D11Buffer* buffer)
	{
		hipError_t cudaStatus = cudaGraphicsD3D11RegisterResource(&mCudaResource, buffer, hipGraphicsRegisterFlagsNone);
		if (cudaStatus != hipSuccess)
		{
			std::cerr << "Direct3D ���ҽ� ��� ����" << std::endl;
			return false;
		}
		return true;
	}

	bool CudaClothPhysics::UpdatePhysicsCloth(physx::PxCudaContextManager* cudaContextManager)
	{
		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &mCudaResource);
		//if (cudaStatus != hipSuccess) {
		//	std::cerr << "hipGraphicsMapResources ����: " << hipGetErrorString(cudaStatus) << std::endl;
		//	return false;
		//}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t size = 0;
		hipGraphicsResourceGetMappedPointer(&devPtr, &size, mCudaResource);
		//if (cudaStatus != hipSuccess) {
		//	std::cerr << "hipGraphicsResourceGetMappedPointer ����: " << hipGetErrorString(cudaStatus) << std::endl;
		//	return false;
		//}

		unsigned int deviceVertexSize = mVertices.size();
		unsigned int deviceIndexSize = mIndices.size();

		physx::PxVec2* d_uvs;
		unsigned int* d_indices;

		hipMalloc(&d_uvs, mUV.size() * sizeof(DirectX::SimpleMath::Vector2));
		hipMalloc(&d_indices, mIndices.size() * sizeof(unsigned int));
		hipMemcpy(d_uvs, mUV.data(), mUV.size() * sizeof(DirectX::SimpleMath::Vector2), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_indices, mIndices.data(), mIndices.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);


		int particleSize = mClothBuffer->getNbActiveParticles();
		physx::PxVec4* particle = mClothBuffer->getPositionInvMasses();

		cudaContextManager->acquireContext();
		physx::PxCudaContext* cudaContext = cudaContextManager->getCudaContext();

		std::vector<physx::PxVec4> vertex;
		vertex.resize(particleSize);

		cudaContext->memcpyDtoH(vertex.data(), hipDeviceptr_t(particle), sizeof(physx::PxVec4) * particleSize);

		for (int i = 0; i < particleSize; i++)
		{
			mVertices[i].x = vertex[i].x;
			mVertices[i].y = vertex[i].y;
			mVertices[i].z = -vertex[i].z;
		}

		int threadsPerBlock = 256;
		int blocksPerGrid = (mIndices.size() / 3 + threadsPerBlock - 1) / threadsPerBlock;

		// CUDA �Լ� ����
		CalculateNormals <<<blocksPerGrid, threadsPerBlock>>> (
			particle, d_uvs, deviceVertexSize, d_indices, deviceIndexSize, (PhysicsVertex*)devPtr);

		hipDeviceSynchronize();

		// CUDA ���ҽ��� �����
		hipGraphicsUnmapResources(1, &mCudaResource);

		// �޸� ����
		hipFree(d_uvs);
		hipFree(d_indices);

		return false;
	}

	void CudaClothPhysics::GetPhysicsCloth(physx::PxCudaContextManager* cudaContextManager, physx::PxCudaContext* cudaContext, PhysicsClothGetData& data)
	{
	}

	void CudaClothPhysics::SetPhysicsCloth(physx::PxCudaContextManager* cudaContextManager, physx::PxCudaContext* cudaContext, const PhysicsClothSetData& data)
	{

	}
}
